#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#include "ggml-common.h"
#include "vecdotq.cuh"
#include "mmq.cuh"


int64_t ggml_get_block_size(int64_t type) {
    switch (type) {
        case GGML_TYPE_Q4_0:    return QK4_0;
        case GGML_TYPE_Q4_1:    return QK4_1;
        case GGML_TYPE_Q5_0:    return QK5_0;
        case GGML_TYPE_Q5_1:    return QK5_1;
        case GGML_TYPE_Q8_0:    return QK8_0;
        case GGML_TYPE_Q8_1:    return QK8_1;
        case GGML_TYPE_Q2_K:    return QK_K;
        case GGML_TYPE_Q3_K:    return QK_K;
        case GGML_TYPE_Q4_K:    return QK_K;
        case GGML_TYPE_Q5_K:    return QK_K;
        case GGML_TYPE_Q6_K:    return QK_K;
        case GGML_TYPE_IQ2_XXS: return QK_K;
        case GGML_TYPE_IQ2_XS:  return QK_K;
        case GGML_TYPE_IQ2_S:   return QK_K;
        case GGML_TYPE_IQ3_XXS: return QK_K;
        case GGML_TYPE_IQ3_S:   return QK_K;
        case GGML_TYPE_IQ1_S:   return QK_K;
        case GGML_TYPE_IQ1_M:   return QK_K;
        case GGML_TYPE_IQ4_NL:  return QK4_NL;
        case GGML_TYPE_IQ4_XS:  return QK_K;
        default: return 0; // unsupported type
    }
}


// Q8 gemv
template <typename scalar_t>
static __global__ void quantize_q8_1(const scalar_t* __restrict__ x,
                                     void* __restrict__ vy, const int kx,
                                     const int kx_padded) {
  const int ix = blockDim.x * blockIdx.x + threadIdx.x;
  if (ix >= kx_padded) {
    return;
  }
  const int iy = blockDim.y * blockIdx.y + threadIdx.y;
  const int i_padded = iy * kx_padded + ix;

  block_q8_1* y = (block_q8_1*)vy;

  const int ib = i_padded / QK8_1;   // block index
  const int iqs = i_padded % QK8_1;  // quant index

  const float xi = ix < kx ? static_cast<float>(x[iy * kx + ix]) : 0.0f;
  float amax = fabsf(xi);
  float sum = xi;

#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    amax = fmaxf(amax, VLLM_SHFL_XOR_SYNC_WIDTH(amax, mask, 32));
    sum += VLLM_SHFL_XOR_SYNC_WIDTH(sum, mask, 32);
  }

  const float d = amax / 127;
  const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

  y[ib].qs[iqs] = q;

  if (iqs > 0) {
    return;
  }

  y[ib].ds.x = __float2half(d);
  y[ib].ds.y = __float2half(sum);
}

template <typename scalar_t>
static void quantize_row_q8_1_cuda(const scalar_t* x, void* vy, const int kx,
                                   const int ky, hipStream_t stream) {
  const int64_t kx_padded = (kx + 512 - 1) / 512 * 512;
  const int block_num_x =
      (kx_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
  constexpr int MAX_BLOCK_SIZE = 65535;
  for (int off = 0; off < ky; off += MAX_BLOCK_SIZE) {
    const int num_blocks_y = std::min(ky, off + MAX_BLOCK_SIZE) - off;
    const dim3 num_blocks(block_num_x, num_blocks_y, 1);
    const dim3 block_size(CUDA_DEQUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(
        &x[off * kx], (int32_t*)vy + off * (kx_padded / 32 * 9), kx, kx_padded);
  }
}


torch::Tensor ggml_mul_mat_a8(torch::Tensor W,  // quant weight
                              torch::Tensor X,  // input
                              int64_t type, int64_t row) {
  int col = X.sizes()[1];
  int padded = (col + 512 - 1) / 512 * 512;
  int batch = X.sizes()[0];
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options = torch::TensorOptions().dtype(X.dtype()).device(W.device());
  at::Tensor Y = torch::empty({batch, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({batch, padded / 32 * 9}, options);
  VLLM_DISPATCH_FLOATING_TYPES(X.scalar_type(), "ggml_mul_mat_a8", [&] {
    quantize_row_q8_1_cuda((scalar_t*)X.data_ptr(), (void*)quant_X.data_ptr(),
                           col, batch, stream);

    const int64_t stride00 = col / ggml_get_block_size(type);
    mmq_args<scalar_t> kernel_args;
    kernel_args = {
        (char*)W.data_ptr(), (char*)quant_X.data_ptr(),
        (scalar_t*)Y.data_ptr(), col, row, stride00, padded, batch, row
    };

    switch (type) {
      case GGML_TYPE_Q4_0:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q4_0>(kernel_args, stream);
        break;
      case GGML_TYPE_Q4_1:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q4_1>(kernel_args, stream);
        break;
      case GGML_TYPE_Q5_0:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q5_0>(kernel_args, stream);
        break;
      case GGML_TYPE_Q5_1:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q5_1>(kernel_args, stream);
        break;
      case GGML_TYPE_Q8_0:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q8_0>(kernel_args, stream);
        break;
      case GGML_TYPE_Q2_K:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q2_K>(kernel_args, stream);
        break;
      case GGML_TYPE_Q3_K:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q3_K>(kernel_args, stream);
        break;
      case GGML_TYPE_Q4_K:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q4_K>(kernel_args, stream);
        break;
      case GGML_TYPE_Q5_K:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q5_K>(kernel_args, stream);
        break;
      case GGML_TYPE_Q6_K:
        mul_mat_q_case<scalar_t, GGML_TYPE_Q6_K>(kernel_args, stream);
        break;
    }
  });
  return Y;
}
